#include "hip/hip_runtime.h"
// This file is part of the AliceVision project.
// This Source Code Form is subject to the terms of the Mozilla Public License,
// v. 2.0. If a copy of the MPL was not distributed with this file,
// You can obtain one at https://mozilla.org/MPL/2.0/.

#include "aliceVision/depthMap/cuda/planeSweeping/cuda_global_data.cuh"

#include "aliceVision/depthMap/cuda/deviceCommon/device_color.cuh"

#include <iostream>

// Macro for checking cuda errors
#define CHECK_CUDA_ERROR()                                                    \
    if(hipError_t err = hipGetLastError())                                  \
                                                                              \
{                                                                             \
        fprintf(stderr, "\n\nCUDAError: %s\n", hipGetErrorString(err));      \
        fprintf(stderr, "  file:       %s\n", __FILE__);                      \
        fprintf(stderr, "  function:   %s\n", __FUNCTION__);                  \
        fprintf(stderr, "  line:       %d\n\n", __LINE__);                    \
                                                                              \
}


namespace aliceVision {
namespace depthMap {

/*
 * We keep data in this array that is frequently allocated and freed, as well
 * as recomputed in the original code without a decent need.
 *
 * The code is not capable of dealing with multiple GPUs yet (on multiple GPUs,
 * multiple allocations are probably required).
 */
GlobalData global_data;

// texture<float, hipTextureType1D, hipReadModeElementType> gaussianTex;

void GaussianArray::create( float delta, int radius )
{
    std::cerr << "Computing Gaussian table for radius " << radius << " and delta " << delta << std::endl;

    int size = 2 * radius + 1;

    float* d_gaussian;
    hipMalloc((void**)&d_gaussian, (2 * radius + 1) * sizeof(float));
    CHECK_CUDA_ERROR();

    // generate gaussian array
    generateGaussian_kernel<<<1, size>>>(d_gaussian, delta, radius);
    hipDeviceSynchronize();

    // create cuda array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipMallocArray(&arr, &channelDesc, size, 1);
    CHECK_CUDA_ERROR();
    hipMemcpyToArray(arr, 0, 0, d_gaussian, size * sizeof(float), hipMemcpyDeviceToDevice);
    CHECK_CUDA_ERROR();
    hipFree(d_gaussian);
    CHECK_CUDA_ERROR();

    hipResourceDesc res_desc;
    res_desc.resType = hipResourceTypeArray;
    res_desc.res.array.array = arr;

    hipTextureDesc      tex_desc;
    memset(&tex_desc, 0, sizeof(hipTextureDesc));
    tex_desc.normalizedCoords = 0; // addressed (x,y) in [width,height]
    tex_desc.addressMode[0]   = hipAddressModeClamp;
    tex_desc.addressMode[1]   = hipAddressModeClamp;
    tex_desc.addressMode[2]   = hipAddressModeClamp;
    tex_desc.readMode         = hipReadModeElementType; // read as float
    tex_desc.filterMode       = hipFilterModePoint; // apparently default for references
    // tex_desc.filterMode       = hipFilterModeLinear; // no interpolation

    hipCreateTextureObject( &tex, &res_desc, &tex_desc, 0 );
    CHECK_CUDA_ERROR();
}

GlobalData::~GlobalData( )
{
    auto end = _gaussian_arr_table.end();
    for( auto it=_gaussian_arr_table.begin(); it!=end;it++ )
    {
        // cudaDestroyTexture( it->second->tex );
        hipFreeArray( it->second->arr );
    }
}

GaussianArray* GlobalData::getGaussianArray( float delta, int radius )
{
    auto it = _gaussian_arr_table.find( GaussianArrayIndex(radius,delta) );
    if( it != _gaussian_arr_table.end() )
    {
        return it->second;
    }

    GaussianArray* a = new GaussianArray;
    a->create( delta, radius );

    _gaussian_arr_table.insert( std::pair<GaussianArrayIndex,GaussianArray*>( GaussianArrayIndex(radius,delta), a ) );

    return a;
}

void GlobalData::allocScaledPictureArrays( int scales, int ncams, int width, int height )
{
    _scaled_picture_scales = scales;

    _scaled_picture_array.resize( scales * ncams );
    _scaled_picture_tex  .resize( scales * ncams );

    hipResourceDesc res_desc;
    res_desc.resType = hipResourceTypeArray;

    hipTextureDesc      tex_desc;
    memset(&tex_desc, 0, sizeof(hipTextureDesc));
    tex_desc.normalizedCoords = 0; // addressed (x,y) in [width,height]
    tex_desc.addressMode[0]   = hipAddressModeClamp;
    tex_desc.addressMode[1]   = hipAddressModeClamp;
    tex_desc.addressMode[2]   = hipAddressModeClamp;
    tex_desc.readMode         = hipReadModeNormalizedFloat;
    tex_desc.filterMode       = hipFilterModeLinear;

    for( int c=0; c<ncams; c++ )
    {
        for( int s=0; s<scales; s++ )
        {
            int w = width / (s + 1);
            int h = height / (s + 1);
            _scaled_picture_array[ c * scales + s ] = new CudaArray<uchar4, 2>( CudaSize<2>( w, h ) );

            res_desc.res.array.array = _scaled_picture_array[ c * scales + s ]->getArray();

            hipCreateTextureObject( &_scaled_picture_tex[ c * scales + s ],
                                     &res_desc,
                                     &tex_desc,
                                     0 );
        }
    }

}

void GlobalData::freeScaledPictureArrays( )
{
    _scaled_picture_scales = 0;

    for( CudaArray<uchar4,2>* ptr : _scaled_picture_array )
    {
        delete ptr;
    }

    _scaled_picture_array.clear();

    for( hipTextureObject_t& obj : _scaled_picture_tex )
    {
        hipDestroyTextureObject( obj );
    }

    _scaled_picture_tex.clear();
}

CudaArray<uchar4,2>* GlobalData::getScaledPictureArrayPtr( int scale, int cam )
{
    return _scaled_picture_array[ cam * _scaled_picture_scales + scale ];
}

CudaArray<uchar4,2>& GlobalData::getScaledPictureArray( int scale, int cam )
{
    return *_scaled_picture_array[ cam * _scaled_picture_scales + scale ];
}

hipTextureObject_t GlobalData::getScaledPictureTex( int scale, int cam )
{
    return _scaled_picture_tex[ cam * _scaled_picture_scales + scale ];
}

void GlobalData::allocPyramidArrays( int levels, int w, int h )
{
    _pyramid_levels = levels;

    _pyramid_array.resize( levels );
    _pyramid_tex  .resize( levels );

    hipTextureDesc      tex_desc;
    memset(&tex_desc, 0, sizeof(hipTextureDesc));
    tex_desc.normalizedCoords = 0; // addressed (x,y) in [width,height]
    tex_desc.addressMode[0]   = hipAddressModeClamp;
    tex_desc.addressMode[1]   = hipAddressModeClamp;
    tex_desc.addressMode[2]   = hipAddressModeClamp;
    tex_desc.readMode         = hipReadModeNormalizedFloat;
    tex_desc.filterMode       = hipFilterModeLinear;

    for( int lvl=0; lvl<levels; lvl++ )
    {
        _pyramid_array[ lvl ] = new CudaDeviceMemoryPitched<uchar4, 2>( CudaSize<2>( w, h ) );

        hipResourceDesc res_desc;
        res_desc.resType = hipResourceTypePitch2D;
        res_desc.res.pitch2D.desc         = hipCreateChannelDesc<uchar4>;
        res_desc.res.pitch2D.devPtr       = _pyramid_array[ lvl ]->getBuffer();
        res_desc.res.pitch2D.width        = _pyramid_array[ lvl ]->getSize()[0];
        res_desc.res.pitch2D.height       = _pyramid_array[ lvl ]->getSize()[1];
        res_desc.res.pitch2D.pitchInBytes = _pyramid_array[ lvl ]->getPitch();

        hipCreateTextureObject( &_pyramid_tex[ lvl ],
                                 &res_desc,
                                 &tex_desc,
                                 0 );
        w /= 2;
        h /= 2;
    }
}

void GlobalData::freePyramidArrays( )
{
    _pyramid_levels = 0;

    for( CudaDeviceMemoryPitched<uchar4,2>* ptr : _pyramid_array )
    {
        delete ptr;
    }

    _pyramid_array.clear();

    for( hipTextureObject_t& obj : _pyramid_tex )
    {
        hipDestroyTextureObject( obj );
    }

    _pyramid_tex.clear();
}

CudaDeviceMemoryPitched<uchar4,2>& GlobalData::getPyramidArray( int level )
{
    return _pyramid_array[ level ];
}

hipTextureObject_t GlobalData::getPyramidTex( int level )
{
    return _pyramid_tex[ level ];
}

}; // namespace depthMap
}; // namespace aliceVision

